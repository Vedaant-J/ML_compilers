#include "hip/hip_runtime.h"
//
// Created by damitha on 1/29/25.
//
#include "utils.h"
#include <hip/hip_runtime.h>

#define CUDA_CHECK(func)                                                     	   \
	do {                                                                           \
		hipError_t status = (func);                                               \
		if (status != hipSuccess) {                                               \
			printf("CUDA API failed at line %d with error: %s (%d)\n", __LINE__,   \
				hipGetErrorString(status), status);                               \
			exit(EXIT_FAILURE);                                                    \
		}                                                                          \
	} while (0)

__global__ void gemm_gpu_kernel(float* A, float* B, float *C, int M, int N, int K) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		for (int i = 0; i < M; i++) {
			for (int j = 0; j < N; j++) {
				C[i * N + j] = 0;
				for (int k = 0; k < K; k++) {
					C[i * N + j]  += A[i * K + k]  * B[k * N + j];
				}
			}
		}
    }
}

void gemm_gpu(float* A, float* B, float* C, int M, int N, int K)
{
	// Init block and grid size
	dim3 blockSize(1);
	dim3 gridSize(1);
	gemm_gpu_kernel<<<gridSize, blockSize>>>(A, B, C, M, N, K);
}

// The scafolding for optimized GEMM implementations
__global__ void gemm_gpu_opt_kernel(float* A, float* B, float *C, int M, int N, int K) {
}
void gemm_gpu_opt(float* A, float* B, float* C, int M, int N, int K)
{
	// Init block and grid size
}


int main(int argc, char* argv[]) {
	if (argc < 3) {
		std::cout << "Usage: mp1 <M> <N> <K>" << std::endl;
		return 1;
	}

	int M = atoi(argv[1]);
	int N = atoi(argv[2]);
	int K = atoi(argv[3]);

	// int runs = atoi(argv[3]);
	float* A = new float[M * K]();
	float* B = new float[K * N]();
	float* C = new float[M * N]();

	fillRandom(A, M * K);
	fillRandom(B, K * N);

	/// GPU Implementation
    // Check if implementation is correct
	float *d_Aref, *d_Bref, *d_Cref;
	CUDA_CHECK(hipMalloc(&d_Aref, Ref::M * Ref::K * sizeof(float)));
	CUDA_CHECK(hipMalloc(&d_Bref, Ref::K * Ref::N * sizeof(float)));
	CUDA_CHECK(hipMalloc(&d_Cref, Ref::M * Ref::N * sizeof(float)));
	auto ref = Ref();
	CUDA_CHECK(hipMemcpy(d_Aref, ref.A, Ref::M * Ref::K * sizeof(float), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(d_Bref, ref.B, Ref::K * Ref::N * sizeof(float), hipMemcpyHostToDevice));
	gemm_gpu(d_Aref, d_Bref, d_Cref, Ref::M, Ref::N, Ref::K);
    // Print errors if there are any
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
	}
	float* refC = new float[Ref::M * Ref::N]();
	CUDA_CHECK(hipMemcpy(refC, d_Cref, Ref::M * Ref::N * sizeof(float), hipMemcpyDeviceToHost));
	if (!ref.checkRef(refC)){
		std::cerr << "check ref failed!" << std::endl;
	};
    // Actual run
	float *d_A, *d_B, *d_C;
	// Device memory allocation
	CUDA_CHECK(hipMalloc(&d_A, M * K * sizeof(float)));
	CUDA_CHECK(hipMalloc(&d_B, K * N * sizeof(float)));
	CUDA_CHECK(hipMalloc(&d_C, M * N * sizeof(float)));
	// Copy host memory to device
	CUDA_CHECK(hipMemcpy(d_A, A, M * K * sizeof(float), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(d_B, B, K * N * sizeof(float), hipMemcpyHostToDevice));
	// For timing
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// Discard the first 5 runs
	for (int i = 0; i < 5; i++)
	{
		gemm_gpu(d_A, d_B, d_C, M, N, K);
	}
	hipError_t err_base = hipGetLastError();
	if (err != hipSuccess) {
		std::cerr << "CUDA Error: " << hipGetErrorString(err_base) << std::endl;
	}
	hipDeviceSynchronize();
	hipEventRecord(start);
	for (int i = 0; i < 100; i++)
	{
		gemm_gpu(d_A, d_B, d_C, M, N, K);
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	// Copy output back to host
	hipMemcpy(C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);
	std::cout << "Time taken for GEMM (GPU, unoptimized): " << milliseconds << "ms" << std::endl;
    // Optimized implementation
	// For timing
	hipEvent_t start_opt, stop_opt;
	hipEventCreate(&start_opt);
	hipEventCreate(&stop_opt);
	// Discard the first 5 runs
	for (int i = 0; i < 5; i++)
	{
		gemm_gpu_opt(A, B, C, M, N, K);
	}
	hipError_t err_opt = hipGetLastError();
	if (err != hipSuccess) {
		std::cerr << "CUDA Error: " << hipGetErrorString(err_opt) << std::endl;
	}
	hipDeviceSynchronize();
	hipEventRecord(start);
	for (int i = 0; i < 100; i++)
	{
		gemm_gpu_opt(A, B, C, M, N, K);
	}
	hipEventRecord(stop_opt);
	hipEventSynchronize(stop_opt);
	float milliseconds_opt = 0;
	hipEventElapsedTime(&milliseconds_opt, start_opt, stop_opt);
	// Copy output back to host
	hipMemcpy(C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);
	std::cout << "Time taken for GEMM (GPU, optimized): " << milliseconds_opt << "ms" << std::endl;


	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipHostFree(A);
	hipHostFree(B);
	hipHostFree(C);

	delete[] A;
	delete[] B;
	delete[] C;

	return 0;
}